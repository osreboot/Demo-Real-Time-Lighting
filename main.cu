#include "hip/hip_runtime.h"
#include <chrono>
#include <glfw/glfw3.h>

#include "display.cuh"
#include "rt_setup.cuh"

using namespace std;

chrono::high_resolution_clock::time_point timeLastUpdate;

int main(){
    display::initialize(1280, 720, "Test");
    rt_setup::initialize();

    timeLastUpdate = chrono::high_resolution_clock::now();

    while(!display::exiting()){
        float delta = (float)chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - timeLastUpdate).count() / 1000.0f;
        delta = min(delta, 0.2f);
        timeLastUpdate = chrono::high_resolution_clock::now();

        display::preUpdate();

        rt_setup::update(delta);

        display::postUpdate();
    }

    glfwTerminate();
    return 0;
}
