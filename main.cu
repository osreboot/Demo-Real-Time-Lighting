#include "hip/hip_runtime.h"
#include <chrono>
#include <glfw/glfw3.h>

#include "config.cuh"
#include "display.cuh"
#include "rt_setup.cuh"

using namespace std;

chrono::high_resolution_clock::time_point timeLastUpdate;

int main(){
    display::initialize(1280, 720, "Real-Time Lighting / Photoreal Demo - by Calvin Weaver");
    rt_setup::initialize();

    bool captured = false;
    timeLastUpdate = chrono::high_resolution_clock::now();

    while(!display::exiting()){
        float delta = (float)chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - timeLastUpdate).count() / 1000.0f;
        delta = min(delta, 0.2f);
        timeLastUpdate = chrono::high_resolution_clock::now();

        display::preUpdate();

        if(PROGRAM_MODE_DYNAMIC){
            rt_setup::update(delta);
        }else if(!captured){
            captured = true;
            rt_setup::capture();
        }

        display::postUpdate();
    }

    glfwTerminate();
    return 0;
}
