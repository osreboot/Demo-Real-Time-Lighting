#include "hip/hip_runtime.h"
#include <chrono>
#include <glfw/glfw3.h>

#include "config.cuh"
#include "display.cuh"
#include "rt_setup.cuh"

using namespace std;

chrono::high_resolution_clock::time_point timeLastUpdate;

int main(){
    // Create the display and initialize the ray tracing program
    display::initialize(1280, 720, "Real-Time Lighting / Photoreal Demo - by Calvin Weaver");
    rt_setup::initialize();

    bool captured = false;
    timeLastUpdate = chrono::high_resolution_clock::now();

    // Loop while the user hasn't clicked the window close button
    while(!display::exiting()){
        // Delta is the time in seconds since the last update. This value is used to synchronize scene timing elements
        // that need to run at a consistent speed, even through inconsistent rendering speeds / update timings (lag).
        float delta = (float)chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - timeLastUpdate).count() / 1000.0f;
        delta = min(delta, 0.2f);
        timeLastUpdate = chrono::high_resolution_clock::now();

        // Prepare the display for rendering
        display::preUpdate();

        // Run the ray tracing program
        if(PROGRAM_MODE_DYNAMIC){
            rt_setup::update(delta);
        }else if(!captured){
            captured = true;
            rt_setup::capture();
        }

        // Render the final image to the display
        display::postUpdate();
    }

    glfwTerminate();
    return 0;
}
