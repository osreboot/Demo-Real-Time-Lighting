#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "config.cuh"
#include "rt_program.cuh"

using namespace owl;

inline __device__ vec3f randomUnitSphere(LCG<4> &random){
    vec3f v;
    do{
        v = 2.0f * vec3f(random(), random(), random()) - vec3f(1.0f, 1.0f, 1.0f);
    }while(dot(v, v) >= 1.0f);
    return v;
}

inline __device__ vec3f tracePath(const RayGenData &self, Ray &ray, PerRayData &prd){
    vec3f attenuation = vec3f(1.0f);
    for(int i = 0; i < 50; i++){
        prd.hitDetected = false;

        traceRay(self.world, ray, prd);

        if(prd.hitCancelled) return vec3f(0.0f);

        attenuation *= prd.color;

        if(!prd.hitDetected) return attenuation;
        ray = Ray(prd.hitOrigin, prd.bounceDirection, 1e-3f, 1e10f);
    }
    return vec3f(0.0f);
}

OPTIX_RAYGEN_PROGRAM(rayGenProgram)(){
    const auto &self = getProgramData<RayGenData>();
    const vec2i pixel = getLaunchIndex();

    // Trace ray
    vec3f color = vec3f(0.0f);
    PerRayData prd;

    for(float i = 0; i < PROGRAM_SAMPLES; i++){
        // Create ray from camera
        Ray ray;
        ray.origin = self.camera.pos;

        const vec2f screen = (vec2f(pixel) + vec2f(prd.random(), prd.random()) + vec2f(0.5f)) / vec2f(self.size);
        ray.direction = normalize(self.camera.dir_00 + screen.u * self.camera.dir_du + screen.v * self.camera.dir_dv);

        vec3f colorOut = tracePath(self, ray, prd);
        colorOut.x = max(min(colorOut.x, 1.0f), 0.0f);
        colorOut.y = max(min(colorOut.y, 1.0f), 0.0f);
        colorOut.z = max(min(colorOut.z, 1.0f), 0.0f);

        color += colorOut;
    }

    // Assign frame buffer pixel color based on ray
    const int indexPixel = pixel.x + self.size.x * pixel.y;
    self.frameBuffer[indexPixel] = make_rgba(color / (float)PROGRAM_SAMPLES);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)(){
    auto &prd = getPRD<PerRayData>();
    auto &self = getProgramData<TrianglesGeomData>();

    // Calculate hitNormal
    const int indexPrimitive = optixGetPrimitiveIndex();
    const vec3i index = self.index[indexPrimitive];
    const vec3f hitNormal = normalize(cross(self.vertex[index.y] - self.vertex[index.x],
                                         self.vertex[index.z] - self.vertex[index.x]));

    const vec3f ro = optixGetWorldRayOrigin();
    const vec3f rd = optixGetWorldRayDirection();
    const float ht = optixGetRayTmax();

    Material &material = self.material[indexPrimitive / 12];

    bool refracted = false;

    prd.hitOrigin = ro + ht * rd;

    // Calculate reflected direction
    vec3f directionReflect = rd - 2.0f * dot(rd, hitNormal) * hitNormal;
    if(prd.random() < material.reflectivity){
        directionReflect = directionReflect + material.diffuse * randomUnitSphere(prd.random);
    }else directionReflect = hitNormal + randomUnitSphere(prd.random);

    // Calculate refracted direction
    vec3f v = normalize(rd);
    float cosine;

    vec3f outward_normal;
    float ni;
    if(dot(v, hitNormal) > 0.0f){
        outward_normal = -hitNormal;
        ni = material.refractiveIndex;
        cosine = dot(v, hitNormal);
        cosine = sqrtf(1.0f - material.refractiveIndex * material.refractiveIndex * (1.0f - cosine * cosine));
    }else{
        outward_normal = hitNormal;
        ni = 1.0f / material.refractiveIndex;
        cosine = -dot(v, hitNormal);
    }

    float dt = dot(v, outward_normal);
    float disc = 1.0f - ni * ni * (1.0f - dt * dt);

    vec3f directionRefract = rd;
    if(disc > 0.0f){
        refracted = true;
        directionRefract = ni * (v - outward_normal * dt) - outward_normal * sqrtf(disc);
    }

    float pReflect = 1.0f;
    if(!refracted){
        pReflect = 1.0f;
    }else{
        // Schlick algorithm
        float r0 = (1.0f - material.refractiveIndex) / (1.0f + material.refractiveIndex);
        r0 = r0 * r0;
        pReflect = r0 + (1.0f - r0) * powf(1.0f - cosine, 5.0f);
    }

    // TODO this isn't actually doing anything
    prd.bounceDirection = directionReflect;
    if(prd.random() < pReflect){ // Reflected
        prd.bounceDirection = directionReflect;
    }else if(prd.random() < material.transparency){ // Refracted
        prd.bounceDirection = directionRefract;
    }

    prd.color = material.color;
    prd.hitDetected = !material.fullbright;
}

OPTIX_MISS_PROGRAM(miss)(){
    auto &prd = getPRD<PerRayData>();

    prd.hitDetected = false;
    prd.hitCancelled = false;

    vec3f rdn = normalize(vec3f(optixGetWorldRayDirection()));
    switch(SCENE_SKYBOX){
        case SCENE_SKYBOX_BLACK:
            prd.color = vec3f(0.0f);
            break;
        case SCENE_SKYBOX_RGB:
            prd.color = (1.0f + rdn) / 2.0f;
            break;
        case SCENE_SKYBOX_UNDERLIT:
            prd.color = 1.0f - max(rdn.y, 0.0f);
            break;
    }
}