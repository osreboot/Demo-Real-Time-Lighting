#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "config.cuh"
#include "rt_program.cuh"

using namespace owl;

inline __device__ vec3f randomUnitSphere(LCG<4>& random){
    vec3f v;
    do{
        v = 2.0f * vec3f(random(), random(), random()) - vec3f(1.0f, 1.0f, 1.0f);
    }while(dot(v, v) >= 1.0f);
    return v;
}

inline __device__ vec3f tracePath(const RayGenData& self, Ray& ray, PerRayData& prd){
    vec3f attenuation = vec3f(1.0f);
    prd.sizeMaterials = 0;
    for(int i = 0; i < 50; i++){
        prd.hitDetected = false;

        traceRay(self.world, ray, prd);

        attenuation *= prd.color;

        if(!prd.hitDetected) return attenuation;

        ray = Ray(prd.hitOrigin, prd.bounceDirection, 1e-3f, 1e10f);
    }
    return vec3f(0.0f);
}

OPTIX_RAYGEN_PROGRAM(rayGenProgram)(){
    const auto &self = getProgramData<RayGenData>();
    const vec2i pixel = getLaunchIndex();

    // Trace ray
    vec3f color = vec3f(0.0f);
    PerRayData prd;

    for(int i = 0; i < PROGRAM_SAMPLES; i++){
        // Create ray from camera
        Ray ray;
        ray.origin = self.camera.pos;

        const vec2f screen = (vec2f(pixel) + vec2f(prd.random(), prd.random()) + vec2f(0.5f)) / vec2f(self.size);
        ray.direction = normalize(self.camera.dir_00 + screen.u * self.camera.dir_du + screen.v * self.camera.dir_dv);

        vec3f colorOut = tracePath(self, ray, prd);
        colorOut.x = max(min(colorOut.x, 1.0f), 0.0f);
        colorOut.y = max(min(colorOut.y, 1.0f), 0.0f);
        colorOut.z = max(min(colorOut.z, 1.0f), 0.0f);

        color += colorOut;
    }

    // Assign frame buffer pixel color based on ray
    const int indexPixel = pixel.x + self.size.x * pixel.y;
    self.frameBuffer[indexPixel] = make_rgba(color / (float)PROGRAM_SAMPLES);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)(){
    auto &prd = getPRD<PerRayData>();
    auto &self = getProgramData<TrianglesGeomData>();

    // Calculate normalSurface
    const unsigned int indexPrimitive = optixGetPrimitiveIndex();
    const vec3i index = self.index[indexPrimitive];
    Material& material = self.material[indexPrimitive / 12];

    const vec3f normalSurface = normalize(cross(self.vertex[index.y] - self.vertex[index.x],
                                                self.vertex[index.z] - self.vertex[index.x]));

    const vec3f ro = optixGetWorldRayOrigin();
    const vec3f rd = optixGetWorldRayDirection();
    const vec3f rdn = normalize(rd);

    prd.hitOrigin = ro + optixGetRayTmax() * rd;

    // Calculate reflected direction
    vec3f directionReflect = rd - 2.0f * dot(rd, normalSurface) * normalSurface;
#if SHADER_LAMBERTIAN_REFLECTION
    if(prd.random() > material.reflectivity){
#if SHADER_SCATTERING
        directionReflect = normalSurface + randomUnitSphere(prd.random);
#else
        directionReflect = normalSurface;
#endif
    }
#endif

    const bool leavingObject = dot(rdn, normalSurface) > 0.0f;

    const Material materialAir = {false, 1.0f, 1.0f, 0.0f, 0.0f, vec3f(1.0f)};

    Material materialLast = leavingObject ? material : (prd.sizeMaterials > 0 ? prd.materials[prd.sizeMaterials - 1] : materialAir);
    Material materialNext = leavingObject ? (prd.sizeMaterials > 1 ? prd.materials[prd.sizeMaterials - 2] : materialAir) : material;

    const bool continuousObject = prd.sizeMaterials > 0 &&
            materialLast.fullbright == materialNext.fullbright &&
            materialLast.transparency == materialNext.transparency &&
            materialLast.refractiveIndex == materialNext.refractiveIndex &&
            materialLast.reflectivity == materialNext.reflectivity &&
            materialLast.diffuse == materialNext.diffuse &&
            materialLast.color == materialNext.color;

    float refractiveIndexRelative = materialNext.refractiveIndex / materialLast.refractiveIndex;

    vec3f normalSurfaceOutwards = leavingObject ? -normalSurface : normalSurface;

    // Algorithm source: https://raytracing.github.io/books/RayTracingInOneWeekend.html#dielectrics/refraction
    float refractCosine = min(dot(-rdn, normalSurfaceOutwards), 1.0f);
#if SHADER_REFRACTION
    vec3f rOutPerp = refractiveIndexRelative * (rdn + refractCosine * normalSurfaceOutwards);
    vec3f rOutPara = -sqrtf(abs(1.0f - length(rOutPerp) * length(rOutPerp))) * normalSurfaceOutwards;
    vec3f directionRefract = rOutPerp + rOutPara;
#else
    vec3f directionRefract = rdn;
#endif

    // Schlick algorithm
#if SHADER_SCHLICK_REFLECTION
    float r0 = (materialLast.refractiveIndex - materialNext.refractiveIndex) / (materialLast.refractiveIndex + materialNext.refractiveIndex);
    r0 = r0 * r0;
    float pReflectSchlick = r0 + (1.0f - r0) * powf(1.0f - refractCosine, 5.0f);
#else
    float pReflectSchlick = 0.0f;
#endif

    if(continuousObject || (prd.random() < materialNext.transparency && prd.random() > pReflectSchlick)){ // Refracted
        if(leavingObject){
            if(prd.sizeMaterials > 0) prd.sizeMaterials--;
        }else{
            if(prd.sizeMaterials < NESTED_MATERIALS_MAX) prd.materials[prd.sizeMaterials++] = material;
        }
        prd.bounceDirection = directionRefract;
    }else prd.bounceDirection = directionReflect;

#if SHADER_SCATTERING
    if(!continuousObject) prd.bounceDirection += material.diffuse * randomUnitSphere(prd.random);
#endif

    prd.color = continuousObject ? vec3f(1.0f) : material.color;

#if SHADER_FULLBRIGHT_MATERIALS
    prd.hitDetected = !material.fullbright;
#else
    prd.hitDetected = true;
#endif
}

OPTIX_MISS_PROGRAM(miss)(){
    auto &prd = getPRD<PerRayData>();

    prd.hitDetected = false;

#if SHADER_SKYBOX_DIRECTIONAL
    vec3f rdn = normalize(vec3f(optixGetWorldRayDirection()));
    switch(SCENE_SKYBOX){
        case SCENE_SKYBOX_BLACK:
            prd.color = vec3f(0.0f);
            break;
        case SCENE_SKYBOX_WHITE:
            prd.color = vec3f(min(1.0f, 1.0f + rdn.y));
            break;
        case SCENE_SKYBOX_RGB:
            prd.color = (1.0f + rdn) / 2.0f;
            break;
        case SCENE_SKYBOX_UNDERLIT:
            prd.color = 1.0f - max(rdn.y, 0.0f);
            break;
    }
#else
    prd.color = vec3f(1.0f);
#endif
}