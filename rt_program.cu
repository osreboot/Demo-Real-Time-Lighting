#include "hip/hip_runtime.h"
#define COUNT_SAMPLES 10.0f

#include <optix_device.h>

#include "rt_program.cuh"

using namespace owl;

inline __device__ vec3f randomUnitSphere(LCG<4> &random){
    vec3f v;
    do{
        v = 2.0f * vec3f(random(), random(), random()) - vec3f(1.0f, 1.0f, 1.0f);
    }while(dot(v, v) >= 1.0f);
    return v;
}

inline __device__ vec3f tracePath(const RayGenData &self, Ray &ray, PerRayData &prd){
    //float attenuation = 1.0f;
    vec3f attenuation = vec3f(1.0f);
    //vec3f color = {1.0f, 1.0f, 1.0f};
    for(int i = 0; i < 50; i++){
        prd.hitDetected = false;

        traceRay(self.world, ray, prd);

        //float mixNew = attenuation * (1.0f - prd.lastTransparency);
        //color = ((1.0f - mixNew) * color) + (mixNew * prd.color);
        //attenuation = prd.lastTransparency * attenuation + (1.0f - prd.lastTransparency) * attenuation * prd.lastReflectivity;

        if(prd.hitCancelled) return vec3f(0.0f);

        attenuation *= prd.color;

        if(!prd.hitDetected) return attenuation;
        ray = Ray(prd.hitOrigin, prd.bounceDirection, 1e-3f, 1e10f);

        /*
        float mixNew = attenuation * (1.0f - prd.lastTransparency);
        color = ((1.0f - mixNew) * color) + (mixNew * prd.color);
        attenuation = prd.lastTransparency * attenuation + (1.0f - prd.lastTransparency) * attenuation * prd.lastReflectivity;

        if(!prd.hitDetected){
            return color;
        }
        if(prd.hitCancelled) return vec3f(0.0f);
        ray = Ray(prd.hitOrigin, prd.bounceDirection, 1e-3f, 1e10f);*/
    }
    return vec3f(0.0f, 0.0f, 0.0f);
}

OPTIX_RAYGEN_PROGRAM(rayGenProgram)(){
    const auto &self = getProgramData<RayGenData>();
    const vec2i pixel = getLaunchIndex();

    // Trace ray
    vec3f color = vec3f(0.0f);
    PerRayData prd;
    //prd.random.init(pixel.x + self.size.x * COUNT_SAMPLES,
    //                pixel.y + self.size.y * COUNT_SAMPLES);

    for(float i = 0; i < COUNT_SAMPLES; i++){
        // Create ray from camera
        Ray ray;
        ray.origin = self.camera.pos;

        const vec2f screen = (vec2f(pixel) + vec2f(prd.random(), prd.random()) + vec2f(0.5f)) / vec2f(self.size);
        ray.direction = normalize(self.camera.dir_00 + screen.u * self.camera.dir_du + screen.v * self.camera.dir_dv);

        vec3f colorOut = tracePath(self, ray, prd);
        colorOut.x = max(min(colorOut.x, 1.0f), 0.0f);
        colorOut.y = max(min(colorOut.y, 1.0f), 0.0f);
        colorOut.z = max(min(colorOut.z, 1.0f), 0.0f);

        color += colorOut;
    }

    // Assign frame buffer pixel color based on ray
    const int indexPixel = pixel.x + self.size.x * pixel.y;
    self.frameBuffer[indexPixel] = make_rgba(color / COUNT_SAMPLES);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)(){
    auto &prd = getPRD<PerRayData>();
    auto &self = getProgramData<TrianglesGeomData>();

    // Calculate normal
    const int indexPrimitive = optixGetPrimitiveIndex();
    const vec3i index = self.index[indexPrimitive];
    const vec3f &a = self.vertex[index.x];
    const vec3f &b = self.vertex[index.y];
    const vec3f &c = self.vertex[index.z];
    const vec3f normal = normalize(cross(b - a, c - a));

    const vec3f ro = optixGetWorldRayOrigin();
    const vec3f rd = optixGetWorldRayDirection();
    const float ht = optixGetRayTmax();

    Material &material = self.material[indexPrimitive / 12];

    bool refracted = false;

    prd.hitDetected = true;
    prd.hitOrigin = ro + ht * rd;

    // Calculate reflected direction
    vec3f directionReflect = rd - 2.0f * dot(rd, normal) * normal;
    if(prd.random() < material.reflectivity){
        directionReflect = directionReflect + material.diffuse * randomUnitSphere(prd.random);
    }else directionReflect = normal + randomUnitSphere(prd.random);

    // Calculate refracted direction
    vec3f v = normalize(rd);
    float cosine;

    // TODO more complicated reflect/refract selection (schlick)
    vec3f outward_normal;
    float ni;
    if(dot(v, normal) > 0.0f){
        outward_normal = -normal;
        ni = material.refractiveIndex;
        cosine = dot(v, normal);
        cosine = sqrtf(1.0f - material.refractiveIndex * material.refractiveIndex * (1.0f - cosine * cosine));
    }else{
        outward_normal = normal;
        ni = 1.0f / material.refractiveIndex;
        cosine = -dot(v, normal);
    }

    float dt = dot(v, outward_normal);
    float disc = 1.0f - ni * ni * (1.0f - dt * dt);

    vec3f directionRefract = rd;
    if(disc > 0.0f){
        refracted = true;
        directionRefract = ni * (v - outward_normal * dt) - outward_normal * sqrtf(disc);
    }

    float pReflect = 1.0f;
    if(!refracted){
        pReflect = 1.0f;
    }else{
        //pReflect = 1.0f - material.transparency;
        // Schlick algorithm
        float r0 = (1.0f - material.refractiveIndex) / (1.0f + material.refractiveIndex);
        r0 = r0 * r0;
        pReflect = r0 + (1.0f - r0) * powf(1.0f - cosine, 5.0f);
    }

    prd.bounceDirection = directionReflect;

    if(prd.random() < pReflect){ // Reflected
        prd.bounceDirection = directionReflect;
    }else if(prd.random() < material.transparency){ // Refracted
        prd.bounceDirection = directionRefract;
    }

    prd.color = material.color;
    prd.hitDetected = !material.fullbright;

    /*
    prd.hitDetected = true;
    prd.hitOrigin = ro + ht * rd;
    if(material.transparency <= 0.0f){
        prd.bounceDirection = rd - 2.0f * dot(rd, normal) * normal;
    }else{
        vec3f v = normalize(rd);

        vec3f outward_normal;
        float ni;
        if(dot(v, normal) > 0.0f){
            outward_normal = -normal;
            ni = material.refractiveIndex;
        }else{
            outward_normal = normal;
            ni = 1.0f / material.refractiveIndex;
        }

        float dt = dot(v, outward_normal);
        float disc = 1.0f - ni * ni * (1.0f - dt * dt);
        if(disc > 0.0f) {
            prd.bounceDirection = ni * (v - outward_normal * dt) - outward_normal * sqrtf(disc);
        }else prd.bounceDirection = rd;
    }

    prd.color = material.color;
    prd.lastReflectivity = material.reflectivity;
    prd.lastTransparency = material.transparency;*/
}

OPTIX_MISS_PROGRAM(miss)(){
    const vec2i pixel = getLaunchIndex();
    auto &prd = getPRD<PerRayData>();

    vec3f rd = normalize(vec3f(optixGetWorldRayDirection()));
    //vec3f colorSkybox = (1.0f + rd) / 2.0f/* + rd.y*/;
    vec3f colorSkybox = 1.0f - max(rd.y, 0.0f);
    //colorSkybox.x = round(colorSkybox.x + 0.3f);
    //colorSkybox.y = round(colorSkybox.y);
    //colorSkybox.z = round(colorSkybox.z + 0.3f);

    prd.hitDetected = false;
    prd.hitCancelled = false;
    //prd.color = vec3f(0.0f);
    prd.color = colorSkybox;
}