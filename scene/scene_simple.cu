#include "hip/hip_runtime.h"
#include "scene.cuh"

#define INDEX_MATERIAL0 3
#define INDEX_MATERIAL1 4
#define INDEX_MATERIAL2 4

class SceneSimple : public Scene{

private:
    const Material MAT_BASE1 = {false, 0.0f, 1.0f, 0.2f, 0.02f, {0.4f, 0.4f, 0.4f}};
    const Material MAT_BASE2 = {false, 0.0f, 1.0f, 0.2f, 0.02f, {0.8f, 0.8f, 0.8f}};

    const Material materials[7] = {
            {true, 0.0f, 1.0f, 0.0f, 0.01f, {1.0f, 1.0f, 0.5f}}, // GLOWING
            {false, 0.0f, 1.0f, 1.0f, 0.01f, {0.6f, 0.0f, 0.0f}}, // SHINY
            {false, 0.0f, 1.0f, 0.0f, 0.0f, {0.0f, 0.6f, 0.0f}}, // LAMBERTIAN
            {false, 0.0f, 1.0f, 1.0f, 0.005f, {0.8f, 0.8f, 0.8f}}, // CHROME
            {false, 0.98f, 2.4f, 1.0f, 0.0f, {0.95f, 0.95f, 1.0f}}, // GLASS
            {false, 0.9f, 1.04f, 1.0f, 0.1f, {0.95f, 1.0f, 1.0f}}, // FROSTED GLASS
            {false, 0.0f, 1.0f, 0.5f, 0.5f, {0.6f, 0.0f, 0.6f}}, // DIELECTRIC
    };

public:
    void initialize(const SceneBuildParams& params) const override {
        int indexTile = 0;
        for(float x = -5.0f; x <= 5.0f; x += 1.0f){
            for(float z = -5.0f; z <= 5.0f; z += 1.0f){
                addBox(params, vec3f(0.5f), {x, -2.5f, z}, 0.0f, indexTile++ % 2 == 0 ? MAT_BASE1 : MAT_BASE2);
            }
        }

        addBox(params, vec3f(0.2f), {0.0f, 0.0f, 0.0f}, 0.0f, materials[INDEX_MATERIAL0]);

        addBox(params, vec3f(1.0f), {0.0f, 0.0f, 0.0f}, 0.0f, materials[INDEX_MATERIAL1]);

        for(vec3f m : {vec3f(1.0f, 1.0f, 1.0f), vec3f(-1.0f, -1.0f, 1.0f), vec3f(1.0f, -1.0f, -1.0f), vec3f(-1.0f, 1.0f, -1.0f)}){
            addBox(params, vec3f(0.6f), m * 0.7f, 0.0f, materials[INDEX_MATERIAL2]);
            addBox(params, vec3f(0.4f), -m * 0.7f, 0.0f, materials[INDEX_MATERIAL2]);
            /*for(vec3f m2 : {vec3f(1.15f, 0.7f, 0.7f), vec3f(0.7f, 1.15f, 0.7f), vec3f(0.7f, 0.7f, 1.15f)}) {
                addBox(params, vec3f(0.2f), m * m2, 0.0f, material);
            }*/
        }

    }

    vec3f getCameraDynamicLocation(const float timer) const override {
        return {sin(timer) * 9.0f, 2.75f, cos(timer) * 9.0f};
    }

    vec3f getCameraDynamicTarget(const float timer) const override {
        return {0.0f, 0.0f, 0.0f};
    }

    vec3f getCameraStaticLocation() const override {
        return vec3f(7.5f, 2.75f, -5.0f) * 1.2f;
    }

    vec3f getCameraStaticTarget() const override {
        return {0.0f, 0.0f, 0.0f};
    }

};