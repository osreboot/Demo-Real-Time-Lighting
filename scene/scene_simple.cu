#include "hip/hip_runtime.h"
#include "scene.cuh"

#define INDEX_MATERIAL 5

class SceneSimple : public Scene{

private:
    const Material MAT_BASE1 = {false, 0.0f, 0.0f, 0.2f, 0.02f, {0.4f, 0.4f, 0.4f}};
    const Material MAT_BASE2 = {false, 0.0f, 0.0f, 0.2f, 0.02f, {0.8f, 0.8f, 0.8f}};

    const Material materials[7] = {
            {true, 0.0f, 0.0f, 0.0f, 0.01f, {1.0f, 1.0f, 0.0f}}, // GLOWING
            {false, 0.0f, 0.0f, 1.0f, 0.01f, {0.6f, 0.0f, 0.0f}}, // SHINY
            {false, 0.0f, 0.0f, 0.0f, 0.0f, {0.0f, 0.6f, 0.0f}}, // LAMBERTIAN
            {false, 0.0f, 0.0f, 1.0f, 0.005f, {0.8f, 0.8f, 0.8f}}, // CHROME
            {false, 0.98f, 1.05f, 1.0f, 0.0f, {0.95f, 0.95f, 1.0f}}, // GLASS
            {false, 0.5f, 1.0f, 1.0f, 0.8f, {0.95f, 1.0f, 1.0f}}, // FROSTED GLASS
            {false, 0.0f, 0.0f, 0.5f, 0.5f, {0.6f, 0.0f, 0.6f}}, // DIELECTRIC
    };

public:
    void initialize(const SceneBuildParams& params) const override {
        int indexTile = 0;
        for(float x = -5.0f; x <= 5.0f; x += 1.0f){
            for(float z = -5.0f; z <= 5.0f; z += 1.0f){
                addBox(params, vec3f(0.5f), {x, -2.5f, z}, 0.0f, indexTile++ % 2 == 0 ? MAT_BASE1 : MAT_BASE2);
            }
        }

        const Material& material = materials[INDEX_MATERIAL];

        addBox(params, vec3f(1.0f), {0.0f, 0.0f, 0.0f}, 0.0f, material);

        for(vec3f m : {vec3f(1.0f, 1.0f, 1.0f), vec3f(-1.0f, -1.0f, 1.0f), vec3f(1.0f, -1.0f, -1.0f), vec3f(-1.0f, 1.0f, -1.0f)}){
            addBox(params, vec3f(0.6f), m * 0.7f, 0.0f, material);
            addBox(params, vec3f(0.4f), -m * 0.7f, 0.0f, material);
            /*for(vec3f m2 : {vec3f(1.15f, 0.7f, 0.7f), vec3f(0.7f, 1.15f, 0.7f), vec3f(0.7f, 0.7f, 1.15f)}) {
                addBox(params, vec3f(0.2f), m * m2, 0.0f, material);
            }*/
        }

    }

    vec3f getCameraDynamicLocation(const float timer) const override {
        return {sin(timer) * 6.0f, 1.75f, cos(timer) * 6.0f};
    }

    vec3f getCameraDynamicTarget(const float timer) const override {
        return {0.0f, 0.0f, 0.0f};
    }

    vec3f getCameraStaticLocation() const override {
        return {5.0f, 1.75f, -3.0f};
    }

    vec3f getCameraStaticTarget() const override {
        return {0.0f, 0.0f, 0.0f};
    }

};