#include "hip/hip_runtime.h"
#include "scene.cuh"

#define INDEX_MATERIAL0 3
#define INDEX_MATERIAL1 4
#define INDEX_MATERIAL2 4

class SceneSimple : public Scene{

private:
    const Material MAT_BASE1 = {false, 0.0f, 1.0f, 0.2f, 0.02f, {0.4f, 0.4f, 0.4f}};
    const Material MAT_BASE2 = {false, 0.0f, 1.0f, 0.2f, 0.02f, {0.8f, 0.8f, 0.8f}};

    const Material materials[7] = {
            {true, 0.0f, 1.0f, 0.0f, 0.01f, {1.0f, 1.0f, 0.5f}}, // GLOWING
            {false, 0.0f, 1.0f, 1.0f, 0.01f, {0.6f, 0.0f, 0.0f}}, // SHINY
            {false, 0.0f, 1.0f, 0.0f, 0.0f, {0.0f, 0.6f, 0.0f}}, // LAMBERTIAN
            {false, 0.0f, 1.0f, 1.0f, 0.005f, {0.8f, 0.8f, 0.8f}}, // CHROME
            {false, 0.98f, 1.5f, 1.0f, 0.0f, {0.95f, 0.95f, 1.0f}}, // GLASS
            {false, 0.9f, 1.04f, 1.0f, 0.2f, {0.9f, 0.9f, 0.9f}}, // FROSTED GLASS
            {false, 0.0f, 1.0f, 0.5f, 0.5f, {0.6f, 0.0f, 0.6f}}, // DIELECTRIC
    };

public:

    // Creates a cube with a bunch of interconnected side cubes and a cube at the core. Change the macro numbers to swap between different materials!
    void initialize(const SceneBuildParams& params) const override {
        int indexTile = 0;
        for(float x = -5.0f; x <= 5.0f; x += 1.0f){
            for(float z = -5.0f; z <= 5.0f; z += 1.0f){
                addBox(params, vec3f(0.5f), {x, -2.5f, z}, 0.0f, indexTile++ % 2 == 0 ? MAT_BASE1 : MAT_BASE2);
            }
        }

        addBox(params, vec3f(0.2f), {0.0f, 0.0f, 0.0f}, 0.0f, materials[INDEX_MATERIAL0]);
        addBox(params, vec3f(1.0f), {0.0f, 0.0f, 0.0f}, 0.0f, materials[INDEX_MATERIAL1]);

        for(vec3f m : {vec3f(1.0f, 1.0f, 1.0f), vec3f(-1.0f, -1.0f, 1.0f), vec3f(1.0f, -1.0f, -1.0f), vec3f(-1.0f, 1.0f, -1.0f)}){
            addBox(params, vec3f(0.6f), m * 0.7f, 0.0f, materials[INDEX_MATERIAL2]);
            addBox(params, vec3f(0.4f), -m * 0.7f, 0.0f, materials[INDEX_MATERIAL2]);
        }

    }

    vec3f getCameraDynamicLocation(const float timer) const override {
        return {sin(timer) * 6.0f, 2.75f, cos(timer) * 6.0f};
    }

    vec3f getCameraDynamicTarget(const float timer) const override {
        return {0.0f, 0.0f, 0.0f};
    }

    vec3f getCameraStaticLocation() const override {
        return vec3f(5.5f, 2.0f, -2.0f) * 0.8f;
    }

    vec3f getCameraStaticTarget() const override {
        return {0.0f, -0.5f, 0.0f};
    }

};