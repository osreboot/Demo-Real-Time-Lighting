#include "hip/hip_runtime.h"
#include "scene.cuh"

#define INDEX_MATERIAL1 1
#define INDEX_MATERIAL2 1

class SceneId : public Scene{

private:
    const Material MAT_BASE1 = {false, 0.0f, 1.0f, 1.0f, 0.01f, {0.5f, 0.5f, 0.8f}};
    const Material MAT_BASE2 = {false, 0.0f, 1.0f, 1.0f, 0.4f, {0.5f, 0.5f, 0.8f}};
    const Material MAT_NUMBERS = {false, 0.0f, 1.0f, 1.0f, 0.005f, {0.8f, 0.8f, 0.8f}};

public:

    // This scene renders my (Calvin Weaver) student ID in big reflective letters with an interesting dual material tiled floor.
    void initialize(const SceneBuildParams& params) const override {
        int indexTile = 0;
        for(float x = -5.0f; x <= 5.0f; x += 1.0f){
            for(float z = -5.0f; z <= 5.0f; z += 1.0f){
                addBox(params, vec3f(0.5f), {x, -0.5f - 0.45f - 0.02f, z}, 0.0f, indexTile++ % 2 == 0 ? MAT_BASE1 : MAT_BASE2);
            }
        }

        const string id = "20007737";
        float x = -((float)id.length() / 2.0f) * 0.7f + 0.35f;
        for(char c : id){
            addBox(params, {0.2f, 0.05f, 0.05f}, {x, 0.4f, 0.0f}, 0.0f, MAT_NUMBERS);
            if(c == '2' || c == '3'){
                addBox(params, {0.2f, 0.05f, 0.05f}, {x, 0.0f, 0.0f}, 0.0f, MAT_NUMBERS);
            }
            if(c == '0' || c == '2' || c == '3'){
                addBox(params, {0.2f, 0.05f, 0.05f}, {x, -0.4f, 0.0f}, 0.0f, MAT_NUMBERS);
            }
            if(c == '0'){
                addBox(params, {0.05f, 0.2f, 0.05f}, {x - 0.25f, 0.25f, 0.0f}, 0.0f, MAT_NUMBERS);
            }
            if(c == '0' || c == '2'){
                addBox(params, {0.05f, 0.25f, 0.05f}, {x - 0.25f, -0.2f, 0.0f}, 0.0f, MAT_NUMBERS);
            }
            if(c == '0' || c == '2' || c == '7' || c == '3'){
                addBox(params, {0.05f, 0.25f, 0.05f}, {x + 0.25f, 0.2f, 0.0f}, 0.0f, MAT_NUMBERS);
            }
            if(c == '0' || c == '7' || c == '3'){
                addBox(params, {0.05f, 0.2f, 0.05f}, {x + 0.25f, -0.25f, 0.0f}, 0.0f, MAT_NUMBERS);
            }
            x += 0.7f;
        }
    }

    vec3f getCameraDynamicLocation(const float timer) const override {
        return {sin(timer) * 6.0f, 2.75f, cos(timer) * 6.0f};
    }

    vec3f getCameraDynamicTarget(const float timer) const override {
        return {0.0f, 0.0f, 0.0f};
    }

    vec3f getCameraStaticLocation() const override {
        return vec3f(3.0f + 0.7f, 2.75f - 0.1f, 5.0f) * 0.6f;
    }

    vec3f getCameraStaticTarget() const override {
        return {0.7f, -0.1f, 0.0f};
    }

};