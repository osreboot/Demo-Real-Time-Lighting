#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "Config.h"
#include "RayTracerDevice.cuh"

using namespace owl;

// Selects a random point inside a unit sphere
inline __device__ vec3f randomUnitSphere(LCG<4>& random) {
    vec3f v;
    do {
        v = 2.0f * vec3f(random(), random(), random()) - vec3f(1.0f, 1.0f, 1.0f);
    } while(dot(v, v) >= 1.0f);
    return v;
}

// Tracks a ray through multiple bounces in the world
inline __device__ vec3f tracePath(const RayGenerator& rayGen, Ray& ray, PerRayData& prd) {
    vec3f attenuation = vec3f(1.0f);
    prd.sizeMaterials = 0;
    // Loop as long as we haven't reached the maximum bounce depth
    for(int i = 0; i < 50; i++){
        prd.hitDetected = false;

        // Launch the ray
        traceRay(rayGen.worldHandle, ray, prd);

        attenuation *= prd.color;

        // The ray hit the sky or a light source
        if(!prd.hitDetected) return attenuation;

        // Re-initialize the ray based on collision parameters
        ray = Ray(prd.hitOrigin, prd.bounceDirection, 1e-3f, 1e10f);
    }

    // Max bounces exceeded
    return {0.0f, 0.0f, 0.0f};
}

// Ray generation program
OPTIX_RAYGEN_PROGRAM(rayGenProgram)() {
    const RayGenerator& rayGen = getProgramData<RayGenerator>();
    const vec2i pixel = getLaunchIndex();

    vec3f color = vec3f(0.0f);
    PerRayData prd;

    // Cast rays to fulfill the number of required samples
    for(int i = 0; i < PROGRAM_SAMPLES; i++){
        // Create ray from camera
        Ray ray;
        ray.origin = rayGen.camera.location;

        // Optional: link the ray's random seed to the pixel position. This is good for static images, but makes
        // real-time renders look like there's dirt on the screen.
        // prd.random.init(pixel.x + self.size.x * i,
        //                 pixel.y + self.size.y * i);

        // Set the ray's position and direction based on the current pixel
        const vec2f screen = (vec2f(pixel) + vec2f(prd.random(), prd.random()) + vec2f(0.5f)) / vec2f(rayGen.size);
        ray.direction = normalize(rayGen.camera.originPixel + screen.u * rayGen.camera.dirRight + screen.v * rayGen.camera.dirUp);

        // Trace the ray's path
        vec3f colorOut = tracePath(rayGen, ray, prd);

        // Clamp the output color
        colorOut.x = max(min(colorOut.x, 1.0f), 0.0f);
        colorOut.y = max(min(colorOut.y, 1.0f), 0.0f);
        colorOut.z = max(min(colorOut.z, 1.0f), 0.0f);

        color += colorOut;
    }

    // Assign frame buffer pixel color based on average of all samples
    rayGen.frameBuffer[pixel.x + rayGen.size.x * pixel.y] = make_rgba(color / (float)PROGRAM_SAMPLES);
}

// Ray hit program
OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)() {
    PerRayData& prd = getPRD<PerRayData>();
    const WorldGeometry& world = getProgramData<WorldGeometry>();

    // Fetch data about the collision surface
    const unsigned int indexPrimitive = optixGetPrimitiveIndex();
    const vec3i index = world.triangles[indexPrimitive];
    const Material& material = world.materials[indexPrimitive / 12];

    // Calculate the normal of the surface
    const vec3f normalSurface = normalize(cross(world.vertices[index.y] - world.vertices[index.x],
                                                world.vertices[index.z] - world.vertices[index.x]));

    const vec3f ro = optixGetWorldRayOrigin();
    const vec3f rd = optixGetWorldRayDirection();
    const vec3f rdn = normalize(rd);

    // Calculate the position of the collision
    prd.hitOrigin = ro + optixGetRayTmax() * rd;

    // Calculate reflected direction
    vec3f directionReflect = rd - 2.0f * dot(rd, normalSurface) * normalSurface;
#if SHADER_LAMBERTIAN_REFLECTION
    if(prd.random() > material.reflectivity){ // Scattering for lambertians
#if SHADER_SCATTERING
        directionReflect = normalSurface + randomUnitSphere(prd.random);
#else
        directionReflect = normalSurface;
#endif
    }
#endif

    // Transparent objects looked REALLY ugly when multiple boxes of the same material were intersecting each other,
    // so this code was added to ignore interior collisions of the same material (look up 3D modeling "boolean union"
    // operation for more context on what exactly this does). Essentially, nested objects of the same material are
    // now treated as one object without interior faces.
    const Material materialAir = {false, 1.0f, 1.0f, 0.0f, 0.0f, vec3f(1.0f)};

    const bool leavingObject = dot(rdn, normalSurface) > 0.0f;
    Material materialLast = leavingObject ? material : (prd.sizeMaterials > 0 ? prd.materials[prd.sizeMaterials - 1] : materialAir);
    Material materialNext = leavingObject ? (prd.sizeMaterials > 1 ? prd.materials[prd.sizeMaterials - 2] : materialAir) : material;

    const bool continuousObject = prd.sizeMaterials > 0 &&
            materialLast.fullbright == materialNext.fullbright &&
            materialLast.transparency == materialNext.transparency &&
            materialLast.refractiveIndex == materialNext.refractiveIndex &&
            materialLast.reflectivity == materialNext.reflectivity &&
            materialLast.diffuse == materialNext.diffuse &&
            materialLast.color == materialNext.color;

    // This refractive index is what actually matters, OWL's sample code doesn't cover this but it's more accurate.
    float refractiveIndexRelative = materialNext.refractiveIndex / materialLast.refractiveIndex;

    vec3f normalSurfaceOutwards = leavingObject ? -normalSurface : normalSurface;

    // Algorithm source: https://raytracing.github.io/books/RayTracingInOneWeekend.html#dielectrics/refraction
    float refractCosine = min(dot(-rdn, normalSurfaceOutwards), 1.0f);
#if SHADER_REFRACTION
    vec3f rOutPerp = refractiveIndexRelative * (rdn + refractCosine * normalSurfaceOutwards);
    vec3f rOutPara = -sqrtf(abs(1.0f - length(rOutPerp) * length(rOutPerp))) * normalSurfaceOutwards;
    vec3f directionRefract = rOutPerp + rOutPara;
#else
    vec3f directionRefract = rdn;
#endif

    // Schlick's approximation for transparent material reflections instead of refractions
#if SHADER_SCHLICK_REFLECTION
    float r0 = (materialLast.refractiveIndex - materialNext.refractiveIndex) / (materialLast.refractiveIndex + materialNext.refractiveIndex);
    r0 = r0 * r0;
    float pReflectSchlick = r0 + (1.0f - r0) * powf(1.0f - refractCosine, 5.0f);
#else
    float pReflectSchlick = 0.0f;
#endif

    // Assign final ray data based on all the above calculations
#if SHADER_SCATTERING
    if(continuousObject || (prd.random() < materialNext.transparency && prd.random() > pReflectSchlick)){ // Refracted
#else
    if(continuousObject || (0.5f < materialNext.transparency && 0.5f > pReflectSchlick)){ // Refracted
#endif
        if(leavingObject){
            if(prd.sizeMaterials > 0) prd.sizeMaterials--;
        }else{
            if(prd.sizeMaterials < NESTED_MATERIALS_MAX) prd.materials[prd.sizeMaterials++] = material;
        }
        prd.bounceDirection = directionRefract;
    }else prd.bounceDirection = directionReflect;

    // Diffuse material scattering
#if SHADER_SCATTERING
    if(!continuousObject) prd.bounceDirection += material.diffuse * randomUnitSphere(prd.random);
#endif

    prd.color = continuousObject ? vec3f(1.0f) : material.color;

#if SHADER_FULLBRIGHT_MATERIALS
    prd.hitDetected = !material.fullbright;
#else
    prd.hitDetected = true;
#endif
}

// Ray miss program
OPTIX_MISS_PROGRAM(miss)() {
    PerRayData& prd = getPRD<PerRayData>();

    prd.hitDetected = false;

#if SHADER_SKYBOX_DIRECTIONAL
    vec3f rayNormal = normalize(vec3f(optixGetWorldRayDirection()));
    switch(SCENE_SKYBOX){
        case SCENE_SKYBOX_BLACK:
            prd.color = vec3f(0.0f);
            break;
        case SCENE_SKYBOX_WHITE:
            prd.color = vec3f(min(1.0f, 1.0f + rayNormal.y));
            break;
        case SCENE_SKYBOX_RGB:
            prd.color = (1.0f + rayNormal) / 2.0f;
            break;
        case SCENE_SKYBOX_UNDERLIT:
            prd.color = 1.0f - max(rayNormal.y, 0.0f);
            break;
        case SCENE_SKYBOX_NIGHT:
            if(rayNormal.y <= 0.0f) prd.color =  vec3f(0.0f);
            else prd.color = vec3f(0.0f, 0.0f, rayNormal.y / 10.0f);
            break;
    }
#else
    prd.color = vec3f(1.0f);
#endif
}