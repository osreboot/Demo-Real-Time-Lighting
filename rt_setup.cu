#include "hip/hip_runtime.h"
#define CAMERA_COS_FOVY 0.66f
#define CAMERA_LOOK_UP vec3f{0.0f, 1.0f, 0.0f}
#define CAMERA_LOOK_AT vec3f{0.0f, 0.0f, 0.0f}

#include <owl/owl.h>
#include <owl/common.h>
#include <vector>
#include <random>

#include "display.cuh"
#include "rt_setup.cuh"
#include "rt_program.cuh"

using namespace std;

extern "C" char rt_program_ptx[];

const int UNIT_NUM_VERTICES = 8;
const vec3f unit_vertices[UNIT_NUM_VERTICES] = {
        {-1.f, -1.f, -1.f},
        {+1.f, -1.f, -1.f},
        {+1.f, +1.f, -1.f},
        {-1.f, +1.f, -1.f},
        {-1.f, +1.f, +1.f},
        {+1.f, +1.f, +1.f},
        {+1.f, -1.f, +1.f},
        {-1.f, -1.f, +1.f}};

const int UNIT_NUM_INDICES = 12;
const vec3i unit_indices[UNIT_NUM_INDICES] = {
        {0, 2, 1}, //face front
        {0, 3, 2},
        {2, 3, 4}, //face top
        {2, 4, 5},
        {1, 2, 5}, //face right
        {1, 5, 6},
        {0, 7, 4}, //face left
        {0, 4, 3},
        {5, 4, 7}, //face back
        {5, 7, 6},
        {0, 6, 7}, //face bottom
        {0, 1, 6}};

OWLRayGen rayGen = {0};
OWLContext context = {0};

vector<vec3f> vertices;
vector<vec3i> indices;
vector<Material> materials;

float timer = 0.0f;

inline float random(){
    static mt19937 gen(0);
    static uniform_real_distribution<float> dist(0.0f, 1.0f);
    return dist(gen);
}

void addBox(vec3f boxSize, vec3f boxLocation, float rotation, Material material){
    affine3f xfm(linear3f::scale(boxSize));
    xfm = affine3f(linear3f::rotate(vec3f(0.0f, 1.0f, 0.0f), ((float)M_PI / 180.0f) * rotation)) * xfm;
    xfm = affine3f(affine3f::translate(boxLocation)) * xfm;

    const int index = (int)vertices.size();
    for(int i = 0; i < UNIT_NUM_VERTICES; i++){
        vertices.push_back(xfmPoint(xfm, unit_vertices[i]));
    }
    for(int i = 0; i < UNIT_NUM_INDICES; i++){
        indices.push_back(unit_indices[i] + index);
    }
    materials.push_back(material);
}

void rt_setup::initialize(){
    const Material matBase1 = {false, 0.0f, 0.0f, 0.5f, 0.02f, {0.4f, 0.4f, 0.4f}};
    const Material matBase2 = {false, 0.0f, 0.0f, 0.5f, 0.2f, {0.8f, 0.8f, 0.8f}};
    const Material matChrome = {false, 0.0f, 0.0f, 1.0f, 0.01f, {0.95f, 0.95f, 0.95f}};
    const Material matBlue = {false, 0.0f, 0.0f, 0.5f, 0.01f, {0.4f, 0.4f, 0.8f}};
    const Material matBlueGlass = {false, 0.5f, 1.33f, 0.5f, 0.0f, {0.4f, 0.4f, 0.8f}};
    const Material matGlass1 = {false, 0.6f, 1.5f, 0.5f, 0.0f, {0.4f, 0.8f, 0.4f}};
    const Material matGlass2 = {false, 0.99f, 1.05f, 0.1f, 0.0f, {1.0f, 1.0f, 1.0f}};
    const Material matFlat1 = {false, 0.0f, 0.0f, 1.0f, 0.5f, {0.5f, 0.5f, 0.0f}};
    const Material matFlat2 = {false, 0.0f, 0.0f, 0.1f, 0.0f, {0.0f, 0.5f, 0.0f}};
    const Material matLight = {true, 0.0f, 0.0f, 0.0f, 0.0f, {1.0f, 1.0f, 1.0f}};
    const Material matLightBlue = {true, 0.0f, 0.0f, 0.0f, 0.0f, {0.0f, 0.0f, 1.0f}};

    int indexTile = 0;
    for(float x = -10.0f; x <= 10.0f; x += 2.0f){
        for(float y = -10.0f; y <= 10.0f; y += 2.0f){
            addBox(vec3f(1.0f), vec3f(x, -1.0f, y), 0.0f, indexTile++ % 2 == 0 ? matBase1 : matBase2);
        }
    }

    addBox(vec3f(0.75f), vec3f(-2.0f, 0.75f, 0.0f), 0.0f, matChrome);
    addBox(vec3f(0.5f), vec3f(2.0f, 0.5f, 3.0f), 45.0f, matChrome);

    addBox(vec3f(6.0f * 0.2f, 0.1f, 6.0f * 0.2f), vec3f(0.0f, 0.1f, -3.0f), 0.0f, matChrome);
    for(float x = -5.0f; x <= 5.0f; x++){
        for(float y = -5.0f; y <= 5.0f; y++){
            addBox(vec3f(0.08f, random() * 0.2f,0.08f), vec3f(x * 0.2f, 0.2f, -3.0f + y * 0.2f), 0.0f, matChrome);
        }
    }

    addBox(vec3f(0.5f, 0.5f, 0.1f), vec3f(0.0f, 0.5f, 3.0f), 0.0f, matChrome);
    addBox(vec3f(0.5f, 0.5f, 0.1f), vec3f(0.0f, 0.5f, 4.0f), 0.0f, matChrome);

    addBox(vec3f(3.0f, 0.1f, 1.0f), vec3f(6.0f, 0.1f, 0.0f), 0.0f, matBlue);
    indexTile = 0;
    for(float x = -2.5f; x <= 2.5f; x += 0.5f){
        addBox(vec3f(0.04f, 0.2f + (x + 2.5f) / 5.0f, 0.4f), vec3f(6.0f + x, 0.3f, cos(x * 1.5f) * 0.5f), sin(x * 1.5f) * 25.0f,
               indexTile++ % 2 == 0 ? matBlue : matBlueGlass);
    }

    addBox(vec3f(0.5f, 3.0f, 0.5f), vec3f(-3.0f, 2.8f, -3.0f), 0.0f, matGlass1);
    addBox(vec3f(0.6f, 0.6f, 0.6f), vec3f(0.0f, 0.8f, 0.0f), 0.0f, matGlass2);
    addBox(vec3f(0.4f, 0.4f, 0.4f), vec3f(-3.0f, 0.6f, 3.0f), 0.0f, matGlass2);
    addBox(vec3f(0.3f, 0.3f, 0.3f), vec3f(-2.8f, 0.3f, 3.2f), 0.0f, matChrome);
    addBox(vec3f(0.3f, 0.3f, 0.3f), vec3f(-3.2f, 0.8f, 2.8f), 0.0f, matChrome);

    addBox(vec3f(0.5f, 0.5f, 0.5f), vec3f(3.0f, 0.7f, -3.0f), 45.0f, matFlat1);
    addBox(vec3f(0.3f, 0.3f, 0.3f), vec3f(3.4f, 0.7f, -3.0f), 45.0f, matFlat1);
    addBox(vec3f(0.3f, 0.3f, 0.3f), vec3f(2.6f, 0.7f, -3.0f), 45.0f, matFlat1);
    addBox(vec3f(1.0f, 0.05f, 1.0f), vec3f(3.0f, 0.05f, -3.0f), 45.0f, matFlat2);

    addBox(vec3f(1.0f, 0.05f, 1.0f), vec3f(3.0f, 2.0f, -3.0f), 45.0f, matLight);
    addBox(vec3f(1.0f, 0.05f, 1.0f), vec3f(-3.0f, 2.0f, 3.0f), 45.0f, matLightBlue);

    context = owlContextCreate(nullptr, 1);
    OWLModule module = owlModuleCreate(context, rt_program_ptx);

    OWLVarDecl trianglesGeomVars[] = {
            {"index", OWL_BUFPTR, OWL_OFFSETOF(TrianglesGeomData, index)},
            {"vertex", OWL_BUFPTR, OWL_OFFSETOF(TrianglesGeomData, vertex)},
            {"material", OWL_BUFPTR, OWL_OFFSETOF(TrianglesGeomData, material)}
    };
    OWLGeomType trianglesGeomType = owlGeomTypeCreate(context, OWL_TRIANGLES, sizeof(TrianglesGeomData), trianglesGeomVars, 3);
    owlGeomTypeSetClosestHit(trianglesGeomType, 0, module, "TriangleMesh");

    OWLBuffer vertexBuffer = owlDeviceBufferCreate(context, OWL_FLOAT3, vertices.size(), vertices.data());
    OWLBuffer indexBuffer = owlDeviceBufferCreate(context, OWL_INT3, indices.size(), indices.data());
    OWLBuffer materialBuffer = owlDeviceBufferCreate(context, OWL_USER_TYPE(materials[0]), materials.size(), materials.data());

    OWLGeom trianglesGeom = owlGeomCreate(context, trianglesGeomType);
    owlTrianglesSetVertices(trianglesGeom, vertexBuffer, vertices.size(), sizeof(vertices[0]), 0);
    owlTrianglesSetIndices(trianglesGeom, indexBuffer, indices.size(), sizeof(indices[0]), 0);

    owlGeomSetBuffer(trianglesGeom, "vertex", vertexBuffer);
    owlGeomSetBuffer(trianglesGeom, "index", indexBuffer);
    owlGeomSetBuffer(trianglesGeom, "material", materialBuffer);

    OWLGroup trianglesGroup = owlTrianglesGeomGroupCreate(context, 1, &trianglesGeom);
    owlGroupBuildAccel(trianglesGroup);
    OWLGroup world = owlInstanceGroupCreate(context, 1, &trianglesGroup);
    owlGroupBuildAccel(world);

    owlMissProgCreate(context, module, "miss", 0, nullptr, 0);

    OWLVarDecl rayGenVars[] = {
            {"frameBuffer", OWL_RAW_POINTER, OWL_OFFSETOF(RayGenData, frameBuffer)},
            {"size", OWL_INT2, OWL_OFFSETOF(RayGenData, size)},
            {"world", OWL_GROUP, OWL_OFFSETOF(RayGenData, world)},
            {"camera.pos", OWL_FLOAT3, OWL_OFFSETOF(RayGenData, camera.pos)},
            {"camera.dir_00", OWL_FLOAT3, OWL_OFFSETOF(RayGenData, camera.dir_00)},
            {"camera.dir_du", OWL_FLOAT3, OWL_OFFSETOF(RayGenData, camera.dir_du)},
            {"camera.dir_dv", OWL_FLOAT3, OWL_OFFSETOF(RayGenData, camera.dir_dv)},
            {}
    };
    rayGen = owlRayGenCreate(context, module, "rayGenProgram", sizeof(RayGenData), rayGenVars, -1);
    owlRayGenSetGroup(rayGen, "world", world);

    owlBuildPrograms(context);
    owlBuildPipeline(context);
}

void rt_setup::update(float delta){
    timer += delta / 2.0f;

    vec3f camera_pos = vec3f(sin(timer) * 10.0f, 4.0f, cos(timer) * 10.0f);
    vec3f camera_d00 = normalize(CAMERA_LOOK_AT - camera_pos);
    float aspect = float(display::getSize().x) / float(display::getSize().y);
    vec3f camera_ddu = CAMERA_COS_FOVY * aspect * normalize(cross(camera_d00,CAMERA_LOOK_UP));
    vec3f camera_ddv = CAMERA_COS_FOVY * normalize(cross(camera_ddu,camera_d00));
    camera_d00 -= 0.5f * camera_ddu;
    camera_d00 -= 0.5f * camera_ddv;

    owlRayGenSet1ul(rayGen, "frameBuffer", (uint64_t)display::getFrameBuffer());
    owlRayGenSet2i(rayGen, "size", display::getSize().x, display::getSize().y);
    owlRayGenSet3f(rayGen, "camera.pos", (const owl3f&)camera_pos);
    owlRayGenSet3f(rayGen, "camera.dir_00", (const owl3f&)camera_d00);
    owlRayGenSet3f(rayGen, "camera.dir_du", (const owl3f&)camera_ddu);
    owlRayGenSet3f(rayGen, "camera.dir_dv", (const owl3f&)camera_ddv);

    owlBuildSBT(context);
    owlRayGenLaunch2D(rayGen, display::getSize().x, display::getSize().y);
}